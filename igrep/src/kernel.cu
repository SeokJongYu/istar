#include "hip/hip_runtime.h"
#include "kernel.hpp"

// About nucleotide.
__constant__ unsigned int *scodon;	/**< The special codon array. */
__constant__ unsigned int character_count;	/**< Number of characters. */
__constant__ unsigned int overlapping_character_count;	/**< Number of overlapping characters between two consecutive threads. */
__constant__ unsigned int overlapping_scodon_count;	/**< Number of overlapping special codons between two consecutive threads. */

// About agrep algorithm.
__constant__ unsigned int       mask_array_32[CHARACTER_CARDINALITY];	/**< The 32-bit mask array of pattern. */
__constant__ unsigned long long mask_array_64[CHARACTER_CARDINALITY];	/**< The 64-bit mask array of pattern. */
__constant__ unsigned int       test_bit_32;	/**< The test bit for determining matches of patterns of length 32. */
__constant__ unsigned long long test_bit_64;	/**< The test bit for determining matches of patterns of length 64. */

// About result.
__constant__ unsigned int max_match_count;	/**< Maximum number of matches of one single query. */
__constant__ unsigned int *match;	/**< The match array. */
__device__ volatile unsigned int match_count;	/**< Number of matches. */

/**
 * The CUDA agrep kernel for matching tables of 32 bits.
 * All the necessary parameters are stored in constant memory.
 */
template<unsigned int KI>
__global__ void agrepKernel32()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[KI + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
	for (k = 1; k <= KI; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
			for (k = 1; k <= KI; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
			for (k = 1; k <= KI; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel for matching tables of 64 bits.
 * All the necessary parameters are stored in constant memory.
 */
template<unsigned int KI>
__global__ void agrepKernel64()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[KI + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
	for (k = 1; k <= KI; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
			for (k = 1; k <= KI; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
			for (k = 1; k <= KI; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
		for (k = 1; k <= KI; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * Transfer necessary parameters to CUDA constant memory.
 * This agrep kernel initialization should be called only once for searching the same corpus.
 * @param[in] scodon_arg The special codon array.
 * @param[in] character_count_arg Actual number of characters.
 * @param[in] match_arg The match array.
 * @param[in] max_match_count_arg Maximum number of matches of one single query.
 */
void initAgrepKernel(const unsigned int *scodon_arg, const unsigned int character_count_arg, const unsigned int *match_arg, const unsigned int max_match_count_arg)
{
	hipMemcpyToSymbol(HIP_SYMBOL(scodon), &scodon_arg, sizeof(unsigned int *));
	hipMemcpyToSymbol(HIP_SYMBOL(character_count), &character_count_arg, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(match), &match_arg, sizeof(unsigned int *));
	hipMemcpyToSymbol(HIP_SYMBOL(max_match_count), &max_match_count_arg, sizeof(unsigned int));
}

/**
 * Transfer 32-bit mask array and test bit from host to CUDA constant memory.
 * @param[in] mask_array_arg The mask array of a pattern.
 * @param[in] test_bit_arg The test bit.
 */
void transferMaskArray32(const unsigned int *mask_array_arg, const unsigned int test_bit_arg)
{
	hipMemcpyToSymbol(HIP_SYMBOL(mask_array_32), mask_array_arg, sizeof(unsigned int) * CHARACTER_CARDINALITY);
	hipMemcpyToSymbol(HIP_SYMBOL(test_bit_32), &test_bit_arg, sizeof(unsigned int));
}

/**
 * Transfer 64-bit mask array and test bit from host to CUDA constant memory.
 * @param[in] mask_array_arg The mask array of a pattern.
 * @param[in] test_bit_arg The test bit.
 */
void transferMaskArray64(const unsigned long long *mask_array_arg, const unsigned long long test_bit_arg)
{
	hipMemcpyToSymbol(HIP_SYMBOL(mask_array_64), mask_array_arg, sizeof(unsigned long long) * CHARACTER_CARDINALITY);
	hipMemcpyToSymbol(HIP_SYMBOL(test_bit_64), &test_bit_arg, sizeof(unsigned long long));
}

/**
 * Invoke the cuda implementation of agrep kernel.
 * @param[in] m Pattern length.
 * @param[in] k Edit distance.
 * @param[in] block_count Number of thread blocks.
 */
void invokeAgrepKernel(const unsigned int m, const unsigned int k, const unsigned int block_count)
{
	unsigned int overlapping_character_count_init = m + k - 1;
	unsigned int overlapping_scodon_count_init = (overlapping_character_count_init + 16 - 1) >> 4;
	unsigned int scodon_header_size = (sizeof(unsigned int) << B) * overlapping_scodon_count_init;	// Used to allocate dynamic shared memory. The first overlapping_scodon_count_init special codons of each thread will be saved into shared memory for the previous thread to continue processing.
	unsigned int match_count_init = 0;

	hipMemcpyToSymbol(HIP_SYMBOL(overlapping_character_count), &overlapping_character_count_init, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(overlapping_scodon_count), &overlapping_scodon_count_init, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(match_count), &match_count_init, sizeof(unsigned int));

	if (m <= 32)
	{
		switch (k)
		{
			case 0:
				agrepKernel32<0><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 1:
				agrepKernel32<1><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 2:
				agrepKernel32<2><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 3:
				agrepKernel32<3><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 4:
				agrepKernel32<4><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 5:
				agrepKernel32<5><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 6:
				agrepKernel32<6><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 7:
				agrepKernel32<7><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 8:
				agrepKernel32<8><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 9:
				agrepKernel32<9><<<block_count, 1 << B, scodon_header_size>>>();
				break;
		}
	}
	else // m > 32
	{
		switch (k)
		{
			case 0:
				agrepKernel64<0><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 1:
				agrepKernel64<1><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 2:
				agrepKernel64<2><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 3:
				agrepKernel64<3><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 4:
				agrepKernel64<4><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 5:
				agrepKernel64<5><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 6:
				agrepKernel64<6><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 7:
				agrepKernel64<7><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 8:
				agrepKernel64<8><<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 9:
				agrepKernel64<9><<<block_count, 1 << B, scodon_header_size>>>();
				break;
		}
	}
}

/**
 * Get the number of matches from CUDA constant memory.
 * @param[out] match_count_arg Number of matches.
 */
void getMatchCount(unsigned int *match_count_arg)
{
	hipMemcpyFromSymbol(match_count_arg, HIP_SYMBOL(match_count), sizeof(unsigned int));
}
