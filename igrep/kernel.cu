
#include <hip/hip_runtime.h>
/*
   Copyright (c) 2011-2012, The Chinese University of Hong Kong

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

/*
   This software contains source code provided by NVIDIA Corporation.
*/

/**
 * @file kernel.cu
 *
 * @brief Here are the CUDA kernels of agrep.
 *
 * @author Hongjian Li
 *
 * Copyright (C) 2011-2012 The Chinese University of Hong Kong.
 */

// About nucleotide.
#define CHARACTER_CARDINALITY 4	/**< One character is either A, C, G, or T. */
__constant__ unsigned int *scodon;	/**< The special codon array. */
__constant__ unsigned int character_count;	/**< Number of characters. */
__constant__ unsigned int overlapping_character_count;	/**< Number of overlapping characters between two consecutive threads. */
__constant__ unsigned int overlapping_scodon_count;	/**< Number of overlapping special codons between two consecutive threads. */

// About agrep algorithm.
__constant__ unsigned int       mask_array_32[CHARACTER_CARDINALITY];	/**< The 32-bit mask array of pattern. */
__constant__ unsigned long long mask_array_64[CHARACTER_CARDINALITY];	/**< The 64-bit mask array of pattern. */
__constant__ unsigned int       test_bit_32;	/**< The test bit for determining matches of patterns of length 32. */
__constant__ unsigned long long test_bit_64;	/**< The test bit for determining matches of patterns of length 64. */

// About result.
__constant__ unsigned int max_match_count;	/**< Maximum number of matches of one single query. */
__constant__ unsigned int *match;	/**< The match array. */
__device__ volatile unsigned int match_count;	/**< Number of matches. */

// About CUDA implementation.
#define MAX_UNSIGNED_INT	0xffffffffUL	/**< The maximum value of an unsigned int. */
#define MAX_UNSIGNED_LONG_LONG	0xffffffffffffffffULL	/**< The maximum value of an unsigned long long. */
#define B 7	/**< Each thread block consists of 2^B (=1<<B) threads. */
#define L 8	/**< Each thread processes 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads. */

// Define constant edit distances so that the entire K+1 matching tables will be saved into registers.
// This trick is extremely essential for boosting performance.
#define K0 0	/**< Constant edit distance 0. */
#define K1 1	/**< Constant edit distance 1. */
#define K2 2	/**< Constant edit distance 2. */
#define K3 3	/**< Constant edit distance 3. */
#define K4 4	/**< Constant edit distance 4. */
#define K5 5	/**< Constant edit distance 5. */
#define K6 6	/**< Constant edit distance 6. */
#define K7 7	/**< Constant edit distance 7. */
#define K8 8	/**< Constant edit distance 8. */
#define K9 9	/**< Constant edit distance 9. */

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 0.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K0()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K0 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K0; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K0; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K0; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 1.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K1()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K1 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K1; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K1; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K1; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 2.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K2()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K2 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K2; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K2; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K2; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 3.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K3()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K3 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K3; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K3; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K3; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 4.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K4()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K4 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K4; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K4; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K4; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 5.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K5()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K5 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K5; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K5; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K5; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 6.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K6()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K6 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K6; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K6; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K6; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 7.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K7()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K7 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K7; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K7; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K7; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 8.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K8()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K8 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K8; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K8; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K8; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 32 bits and edit distance of 9.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel32K9()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned int mask_word;	// The mask word of a character from mask array.
	unsigned int r[K9 + 1];	// The most recent columns of K+1 matching tables.
	unsigned int r0;	// The second most recent column of previous matching table.
	unsigned int r1;	// The        most recent column of previous matching table.
	unsigned int r2;	// The second most recent column of current  matching table.
	unsigned int r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_INT;
#pragma unroll
	for (k = 1; k <= K9; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K9; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_32[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_32[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K9; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_32) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_32[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_32) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 0.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K0()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K0 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K0; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K0; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K0; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K0; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 1.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K1()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K1 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K1; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K1; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K1; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K1; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 2.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K2()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K2 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K2; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K2; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K2; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K2; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 3.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K3()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K3 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K3; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K3; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K3; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K3; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 4.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K4()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K4 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K4; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K4; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K4; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K4; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 5.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K5()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K5 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K5; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K5; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K5; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K5; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 6.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K6()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K6 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K6; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K6; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K6; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K6; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 7.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K7()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K7 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K7; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K7; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K7; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K7; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 8.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K8()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K8 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K8; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K8; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K8; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K8; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * The CUDA agrep kernel with matching tables of 64 bits and edit distance of 9.
 * All the necessary parameters are stored in constant memory.
 */
__global__ void agrepKernel64K9()
{
	// About CUDA implementation.
	extern __shared__ unsigned int scodon_header[][1 << B];	// Used to store the first overlapping_scodon_count special codons of each thread of a thread block.
	unsigned int block_base_index;	// The base index of current thread block.
	unsigned int inputting_scodon_base_index;	// The base index into inputting  special codon of current thread.
	unsigned int scodon_index;	// Used to enumerate the 2^L (=1<<L) special codons plus those in the overlapping zone of two consecutive threads.
	unsigned int scodon_buffer;	// The special codon currently being processed.

	// About agrep algorithm.
	unsigned int character_index;	// Used to enumerate the special codon buffer.
	unsigned long long mask_word;	// The mask word of a character from mask array.
	unsigned long long r[K9 + 1];	// The most recent columns of K+1 matching tables.
	unsigned long long r0;	// The second most recent column of previous matching table.
	unsigned long long r1;	// The        most recent column of previous matching table.
	unsigned long long r2;	// The second most recent column of current  matching table.
	unsigned long long r3;	// The        most recent column of current  matching table. r3 = function(r0, r1, r2, mask_value);
	unsigned int k;		// Used to enumerate K+1 matching tables.

	// About result.
	unsigned int outputting_scodon_base_index;	// The base index into outputting special codon of current thread.
	unsigned int matching_character_index;	// The output of the kernel. It stores the matching ending position.

	block_base_index = blockIdx.x << (L + B);	// The base index of current thread block.
	inputting_scodon_base_index  = block_base_index + threadIdx.x;	// Coalesced global memory access is ensured.
	outputting_scodon_base_index = block_base_index + (threadIdx.x << L);	// Original order of corpus.
	r[0] = MAX_UNSIGNED_LONG_LONG;
#pragma unroll
	for (k = 1; k <= K9; k++)
		r[k] = r[k - 1] << 1;	// Initialize K+1 matching tables according to agrep algorithm.
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K9; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
		}
		scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	}
	scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
	}
	for (; character_index < 16; character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			/* A possible match is found.
			 *   1) Calculate the matching character index, and ensure it does not exceed the corpus boundary.
			 *   2) Atomically increase match_count by 1, whose original value points to the index that the current match should be saved at.
			 *   3) Save the matching character index to the match array, if the max number of matches has not yet been exceeded.
			 */
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}		
	}
	scodon_header[scodon_index][threadIdx.x] = scodon_buffer;
	__syncthreads();	// Make sure all the threads of current thread block have saved their first overlapping_scodon_count special codons to the shared memory for later use by the previous thread.
	for (scodon_index++; scodon_index < (1 << L); scodon_index++)	// These special codons at index [overlapping_scodon_count, 2^L) are processed by current thread only once, hence no need to save them into shared memory.
	{
		scodon_buffer = scodon[inputting_scodon_base_index + (scodon_index << B)];
		mask_word = mask_array_64[(scodon_buffer >> 0) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 0;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 2) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 1;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 4) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 2;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 6) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 3;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 8) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 4;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 10) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 5;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 12) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 6;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 14) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 7;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 16) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 8;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 18) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 9;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 20) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 10;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 22) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 11;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 24) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 12;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 26) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 13;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 28) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 14;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
		mask_word = mask_array_64[(scodon_buffer >> 30) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + scodon_index) << 4) + 15;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
	for (scodon_index = 0; scodon_index < overlapping_scodon_count - 1; scodon_index++)
	{
		scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
		for (character_index = 0; character_index < 16; character_index++)
		{
			mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
			r2 = r[0];
			r3 = (r2 << 1) | mask_word;
			r[0] = r3;
#pragma unroll
			for (k = 1; k <= K9; k++)
			{
				r0 = r2;
				r1 = r3;
				r2 = r[k];
				r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
				r[k] = r3;
			}
			if (!(r3 & test_bit_64) && (match_count < max_match_count))
			{
				matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
				if (matching_character_index <= character_count)
					match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
			}
		}
	}
	scodon_buffer = ((threadIdx.x == (blockDim.x - 1)) ? scodon[block_base_index + (1 << (L + B)) + (scodon_index << B)] : scodon_header[scodon_index][threadIdx.x + 1]);
	for (character_index = 0; character_index < overlapping_character_count - ((overlapping_scodon_count - 1) << 4); character_index++)
	{
		mask_word = mask_array_64[(scodon_buffer >> (character_index << 1)) & 3];
		r2 = r[0];
		r3 = (r2 << 1) | mask_word;
		r[0] = r3;
#pragma unroll
		for (k = 1; k <= K9; k++)
		{
			r0 = r2;
			r1 = r3;
			r2 = r[k];
			r3 = ((r2 << 1) | mask_word) & ((r0 & r1) << 1) & r0;
			r[k] = r3;
		}
		if (!(r3 & test_bit_64) && (match_count < max_match_count))
		{
			matching_character_index = ((outputting_scodon_base_index + (1 << L) + scodon_index) << 4) + character_index;
			if (matching_character_index <= character_count)
				match[atomicAdd((unsigned int *)&match_count, 1)] = matching_character_index;
		}
	}
}

/**
 * Transfer necessary parameters to CUDA constant memory.
 * This agrep kernel initialization should be called only once for searching the same corpus.
 * @param[in] scodon_arg The special codon array.
 * @param[in] character_count_arg Actual number of characters.
 * @param[in] match_arg The match array.
 * @param[in] max_match_count_arg Maximum number of matches of one single query.
 */
extern "C" void initAgrepKernel(const unsigned int *scodon_arg, const unsigned int character_count_arg, const unsigned int *match_arg, const unsigned int max_match_count_arg)
{
	hipMemcpyToSymbol(HIP_SYMBOL(scodon), &scodon_arg, sizeof(unsigned int *));
	hipMemcpyToSymbol(HIP_SYMBOL(character_count), &character_count_arg, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(match), &match_arg, sizeof(unsigned int *));
	hipMemcpyToSymbol(HIP_SYMBOL(max_match_count), &max_match_count_arg, sizeof(unsigned int));
}

/**
 * Transfer 32-bit mask array and test bit from host to CUDA constant memory.
 * @param[in] mask_array_arg The mask array of a pattern.
 * @param[in] test_bit_arg The test bit.
 */
extern "C" void transferMaskArray32(const unsigned int *mask_array_arg, const unsigned int test_bit_arg)
{
	hipMemcpyToSymbol(HIP_SYMBOL(mask_array_32), mask_array_arg, sizeof(unsigned int) * CHARACTER_CARDINALITY);
	hipMemcpyToSymbol(HIP_SYMBOL(test_bit_32), &test_bit_arg, sizeof(unsigned int));
}

/**
 * Transfer 64-bit mask array and test bit from host to CUDA constant memory.
 * @param[in] mask_array_arg The mask array of a pattern.
 * @param[in] test_bit_arg The test bit.
 */
extern "C" void transferMaskArray64(const unsigned long long *mask_array_arg, const unsigned long long test_bit_arg)
{
	hipMemcpyToSymbol(HIP_SYMBOL(mask_array_64), mask_array_arg, sizeof(unsigned long long) * CHARACTER_CARDINALITY);
	hipMemcpyToSymbol(HIP_SYMBOL(test_bit_64), &test_bit_arg, sizeof(unsigned long long));
}

/**
 * Invoke the cuda implementation of agrep kernel.
 * @param[in] m Pattern length.
 * @param[in] k Edit distance.
 * @param[in] block_count Number of thread blocks.
 */
extern "C" void invokeAgrepKernel(const unsigned int m, const unsigned int k, const unsigned int block_count)
{
	unsigned int overlapping_character_count_init = m + k - 1;
	unsigned int overlapping_scodon_count_init = (overlapping_character_count_init + 16 - 1) >> 4;
	unsigned int scodon_header_size = (sizeof(unsigned int) << B) * overlapping_scodon_count_init;	// Used to allocate dynamic shared memory. The first overlapping_scodon_count_init special codons of each thread will be saved into shared memory for the previous thread to continue processing.
	unsigned int match_count_init = 0;

	hipMemcpyToSymbol(HIP_SYMBOL(overlapping_character_count), &overlapping_character_count_init, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(overlapping_scodon_count), &overlapping_scodon_count_init, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(match_count), &match_count_init, sizeof(unsigned int));

	if (m <= 32)
	{
		switch (k)
		{
			case 0:
				agrepKernel32K0<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 1:
				agrepKernel32K1<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 2:
				agrepKernel32K2<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 3:
				agrepKernel32K3<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 4:
				agrepKernel32K4<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 5:
				agrepKernel32K5<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 6:
				agrepKernel32K6<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 7:
				agrepKernel32K7<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 8:
				agrepKernel32K8<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 9:
				agrepKernel32K9<<<block_count, 1 << B, scodon_header_size>>>();
				break;
		}
	}
	else // m > 32
	{
		switch (k)
		{
			case 0:
				agrepKernel64K0<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 1:
				agrepKernel64K1<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 2:
				agrepKernel64K2<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 3:
				agrepKernel64K3<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 4:
				agrepKernel64K4<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 5:
				agrepKernel64K5<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 6:
				agrepKernel64K6<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 7:
				agrepKernel64K7<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 8:
				agrepKernel64K8<<<block_count, 1 << B, scodon_header_size>>>();
				break;
			case 9:
				agrepKernel64K9<<<block_count, 1 << B, scodon_header_size>>>();
				break;
		}
	}
}

/**
 * Get the number of matches from CUDA constant memory.
 * @param[out] match_count_arg Number of matches.
 */
extern "C" void getMatchCount(unsigned int *match_count_arg)
{
	hipMemcpyFromSymbol(match_count_arg, HIP_SYMBOL(match_count), sizeof(unsigned int));
}